#include "hip/hip_runtime.h"
/**
FNELEM-GPU GPU MATRIX INVERSION
Performs matrix inversion using Gauss Jordan algorithm.
Based on: https://github.com/ZhengzhongSun/Matrix-Inversion-with-CUDA

@package fnelem.math
@author ppizarror
@date 19/11/2018
@license
	MIT License
	Copyright (c) 2018 Pablo Pizarro R.

	Permission is hereby granted, free of charge, to any person obtaining a copy
	of this software and associated documentation files (the "Software"), to deal
	in the Software without restriction, including without limitation the rights
	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
	copies of the Software, and to permit persons to whom the Software is
	furnished to do so, subject to the following conditions:

	The above copyright notice and this permission notice shall be included in all
	copies or substantial portions of the Software.

	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
	SOFTWARE.
*/

// Library imports
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "FEMatrix.h"

// Constants
const int MATRIX_INVERSION_CUDA_BLOCKSIZE = 8;

/**
 * NODIAG normalize diagonal matrix (CUDA).
 *
 * @param A Matrix
 * @param I Matrix
 * @param n Dimension
 * @param i Position
 */
__global__ void nodiag_normalize(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n)
        if (x == i && x != y) {
            I[x * n + y] /= A[i * n + i];
            A[x * n + y] /= A[i * n + i];
        }
}

/**
 * DIAG normalize diagonal matrix (CUDA).
 *
 * @param A Matrix
 * @param I Matrix
 * @param n Dimension
 * @param i Position
 */
__global__ void diag_normalize(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n)
        if (x == y && x == i) {
            I[x * n + y] /= A[i * n + i];
            A[x * n + y] /= A[i * n + i];
        }
}

/**
 * Performs Gauss Jordan algorithm (CUDA).
 *
 * @param A Matrix
 * @param I Matrix
 * @param n Dimension
 * @param i Position
 */
__global__ void gaussjordan(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n) {
        if (x != i) {
            I[x * n + y] -= I[i * n + y] * A[x * n + i];
            if (y != i) {
                A[x * n + y] -= A[i * n + y] * A[x * n + i];
            }
        }
    }
}

/**
 * Set zero on matrix (CUDA).
 *
 * @param A Matrix
 * @param I Matrix
 * @param n Dimension
 * @param i Position
 */
__global__ void set_zero(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n) {
        if (x != i) {
            if (y == i) {
                A[x * n + y] = 0;
            }
        }
    }
}

/**
 * Save matrix to file.
 *
 * @param A Matrix
 * @param s File name
 * @param n Number of rows
 * @param h Number of columns
 */
void save_matrix_to_file(double *A, std::string s, int n, int h) {
    std::ofstream plik;
    plik.open(s);
    for (int j = 0; j < h; j++) {
        for (int i = 0; i < h; i++) {
            plik << A[j * n + i] << "\t";
        }
        plik << std::endl;
    }
    plik.close();
}

/**
 * Matrix inversion, uses CUDA.
 *
 * @param feMatrix Matrix to inverse
 * @return Inverse matrix
 */
FEMatrix *matrix_inverse_cuda(FEMatrix *feMatrix) {

    // Get matrix
    double *matrix = feMatrix->get_array();

    // Get matrix dimension
    int *matDim = feMatrix->get_dimension();
    int n;
    if (matDim[0] == matDim[1]) {
        n = matDim[0];
    } else {
        throw std::logic_error("Matrix to inverse is not square");
    }

    // Inverse matrix CPU
    double *iMatrix = new double[n * n];

    // Create auxiliar matrices
    double *d_A, *I, *dI;

    // Time of computation
    float time;

    // Create CUDA error handlers
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Matrix memory size
    int ddsize = n * n * sizeof(double);

    // Creates blocks
    dim3 threadsPerBlock(MATRIX_INVERSION_CUDA_BLOCKSIZE, MATRIX_INVERSION_CUDA_BLOCKSIZE);
    dim3 numBlocks((n + MATRIX_INVERSION_CUDA_BLOCKSIZE - 1) / MATRIX_INVERSION_CUDA_BLOCKSIZE,
                   (n + MATRIX_INVERSION_CUDA_BLOCKSIZE - 1) / MATRIX_INVERSION_CUDA_BLOCKSIZE);

    // Memory allocation
    err = hipMalloc((void **) &d_A, ddsize);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl;
    }
    err = hipMalloc((void **) &dI, ddsize);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl;
    }

    // Creates identify matrix
    I = new double[n * n];
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) I[i * n + i] = 1.0;
            else I[i * n + j] = 0.0;
        }
    }

    // Copy data from CPU to GPU
    err = hipMemcpy(d_A, matrix, ddsize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl;
    }
    err = hipMemcpy(dI, I, ddsize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl;
    }

    // Timer start
    hipEventRecord(start, 0);

    // L^(-1)
    for (int i = 0; i < n; i++) {
        nodiag_normalize << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        diag_normalize << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        gaussjordan << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        set_zero << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
    }

    // Record cuda events
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy data from GPU to CPU
    err = hipMemcpy(iMatrix, dI, ddsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl;
    }
    err = hipMemcpy(I, d_A, ddsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << std::endl;
    }
    std::cout << "[CUDA] Matrix inversion time: " << time << "ms\n" << std::endl;

    // Free memory
    hipFree(d_A);
    hipFree(dI);
    delete[] I;

    // Generate matrix
    return new FEMatrix(iMatrix, n, n);

}