#include "hip/hip_runtime.h"
/**
FNELEM-GPU MATRIX INVERSION

Performs matrix inversion using Gauss Jordan algorithm.
Based on: https://github.com/ZhengzhongSun/Matrix-Inversion-with-CUDA

@package fnelem.analysis
@author ppizarror
@date 19/11/2018
@license
    MIT License
    Copyright (c) 2018 Pablo Pizarro R.

    Permission is hereby granted, free of charge, to any person obtaining a copy
    of this software and associated documentation files (the "Software"), to deal
    in the Software without restriction, including without limitation the rights
    to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
    copies of the Software, and to permit persons to whom the Software is
    furnished to do so, subject to the following conditions:

    The above copyright notice and this permission notice shall be included in all
    copies or substantial portions of the Software.

    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
    AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
    OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
    SOFTWARE.
*/

// Library imports
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <string>
#include <vector>

/**
 * NODIAG normalize diagonal matrix (CUDA).
 *
 * @param A Matrix
 * @param I Matrix
 * @param n Dimension
 * @param i Position
 */
__global__ void nodiag_normalize(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n)
        if (x == i && x != y) {
            I[x * n + y] /= A[i * n + i];
            A[x * n + y] /= A[i * n + i];
        }
}

/**
 * DIAG normalize diagonal matrix (CUDA).
 *
 * @param A Matrix
 * @param I Matrix
 * @param n Dimension
 * @param i Position
 */
__global__ void diag_normalize(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n)
        if (x == y && x == i) {
            I[x * n + y] /= A[i * n + i];
            A[x * n + y] /= A[i * n + i];
        }
}

/**
 * Performs Gauss Jordan algorithm (CUDA).
 *
 * @param A Matrix
 * @param I Matrix
 * @param n Dimension
 * @param i Position
 */
__global__ void gaussjordan(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n) {
        if (x != i) {
            I[x * n + y] -= I[i * n + y] * A[x * n + i];
            if (y != i) {
                A[x * n + y] -= A[i * n + y] * A[x * n + i];
            }
        }
    }
}

/**
 * Set zero on matrix (CUDA).
 *
 * @param A Matrix
 * @param I Matrix
 * @param n Dimension
 * @param i Position
 */
__global__ void set_zero(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n) {
        if (x != i) {
            if (y == i) {
                A[x * n + y] = 0;
            }
        }
    }
}

/**
 * Matrix inversion, uses CUDA.
 *
 * @param matrix Matrix to inverse
 * @param n Matrix dimension
 * @return Inverse matrix
 */
double *inverse_matrix(double *matrix, int n) {

    // Creates matrices
    double *iL = new double[n * n];
    double *L = new double[n * n];

    L[0 * 3 + 0] = 1;
    L[0 * 3 + 1] = 2;
    L[0 * 3 + 2] = 3;
    L[1 * 3 + 0] = 5;
    L[1 * 3 + 1] = 2;
    L[1 * 3 + 2] = 1;
    L[2 * 3 + 0] = 2;
    L[2 * 3 + 1] = 2;
    L[2 * 3 + 2] = 3;

    // matrix_read(L, n);
    save_inverse_matrix_to_file(L, "inv4.txt", n, n);
    //savetofile(L, "L.txt", n, n);

    double *d_A, *d_L, *I, *dI;
    float time;
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int ddsize = n * n * sizeof(double);
    int blocksize = 8;

    dim3 threadsPerBlock(blocksize, blocksize);
    dim3 numBlocks((n + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);
    // memory allocation
    err = hipMalloc((void **) &d_A, ddsize);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }
    err = hipMalloc((void **) &dI, ddsize);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }
    I = new double[n * n];

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) I[i * n + i] = 1.0;
            else I[i * n + j] = 0.0;
        }
    }

    //copy data from CPU to GPU
    err = hipMemcpy(d_A, L, ddsize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }
    err = hipMemcpy(dI, I, ddsize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }

    //timer start
    hipEventRecord(start, 0);

    // L^(-1)
    for (int i = 0; i < n; i++) {
        nodiag_normalize << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        diag_normalize << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        gaussjordan << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        set_zero << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //copy data from GPU to CPU
    err = hipMemcpy(iL, dI, ddsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }
    err = hipMemcpy(I, d_A, ddsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }

    cout << "Cuda Time - inverse: " << time << "ms\n";
    save_inverse_matrix_to_file(iL, "inv1.txt", n, n);
    save_inverse_matrix_to_file(I, "inv2.txt", n, n);
    save_inverse_matrix_to_file(L, "inv3.txt", n, n);
    //savetofile(I, "I.txt", n, n);
    //savetofile(I, "I.txt", n, n);
    hipFree(d_A);
    hipFree(dI);

    double *c = new double[n * n];
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            c[i * n + j] = 0;  //put the initial value to zero
            for (int x = 0; x < n; x++)
                c[i * n + j] = c[i * n + j] + L[i * n + x] * iL[x * n + j];  //matrix multiplication
        }
    save_inverse_matrix_to_file(c, "c.txt", n, n);

    delete[]I;
    delete[]L;
    delete[]iL;
}