/**
FNELEM-GPU MAIN FILE
Performs finite element structural analysis using an 4-node membrane, matrix inversion
was calculated using a CUDA algorithm (Gauss Jordan inversion).

@package fnelem.analysis
@author ppizarror
@date 19/11/2018
@license
    MIT License
    Copyright (c) 2018 Pablo Pizarro R.

    Permission is hereby granted, free of charge, to any person obtaining a copy
    of this software and associated documentation files (the "Software"), to deal
    in the Software without restriction, including without limitation the rights
    to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
    copies of the Software, and to permit persons to whom the Software is
    furnished to do so, subject to the following conditions:

    The above copyright notice and this permission notice shall be included in all
    copies or substantial portions of the Software.

    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
    AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
    OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
    SOFTWARE.
*/

// CUDA library imports
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// FNELEM library imports
#include "fnelem/analysis/matrix_inversion.cu"

int main() {

    const int n = 3;
    // creating input
    double *iL = new double[n * n];
    double *L = new double[n * n];

    L[0 * 3 + 0] = 1;
    L[0 * 3 + 1] = 2;
    L[0 * 3 + 2] = 3;
    L[1 * 3 + 0] = 5;
    L[1 * 3 + 1] = 2;
    L[1 * 3 + 2] = 1;
    L[2 * 3 + 0] = 2;
    L[2 * 3 + 1] = 2;
    L[2 * 3 + 2] = 3;

    // matrix_read(L, n);
    save_inverse_matrix_to_file(L, "inv4.txt", n, n);
    //savetofile(L, "L.txt", n, n);

    cout << "inv\n";
    double *d_A, *d_L, *I, *dI;
    float time;
    hipError_t err;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int ddsize = n * n * sizeof(double);
    int blocksize = 8;

    dim3 threadsPerBlock(blocksize, blocksize);
    dim3 numBlocks((n + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);
    // memory allocation
    err = hipMalloc((void **) &d_A, ddsize);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }
    err = hipMalloc((void **) &dI, ddsize);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }
    I = new double[n * n];

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i == j) I[i * n + i] = 1.0;
            else I[i * n + j] = 0.0;
        }
    }

    //copy data from CPU to GPU
    err = hipMemcpy(d_A, L, ddsize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }
    err = hipMemcpy(dI, I, ddsize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }

    //timer start
    hipEventRecord(start, 0);

    // L^(-1)
    for (int i = 0; i < n; i++) {
        nodiag_normalize << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        diag_normalize << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        gaussjordan << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
        set_zero << < numBlocks, threadsPerBlock >> > (d_A, dI, n, i);
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //copy data from GPU to CPU
    err = hipMemcpy(iL, dI, ddsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }
    err = hipMemcpy(I, d_A, ddsize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl;
    }

    cout << "Cuda Time - inverse: " << time << "ms\n";
    save_inverse_matrix_to_file(iL, "inv1.txt", n, n);
    save_inverse_matrix_to_file(I, "inv2.txt", n, n);
    save_inverse_matrix_to_file(L, "inv3.txt", n, n);
    //savetofile(I, "I.txt", n, n);
    //savetofile(I, "I.txt", n, n);
    hipFree(d_A);
    hipFree(dI);

    double *c = new double[n * n];
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) {
            c[i * n + j] = 0;  //put the initial value to zero
            for (int x = 0; x < n; x++)
                c[i * n + j] = c[i * n + j] + L[i * n + x] * iL[x * n + j];  //matrix multiplication
        }
    save_inverse_matrix_to_file(c, "c.txt", n, n);

    delete[]I;
    delete[]L;
    delete[]iL;













    /**
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
           c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size) {
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **) &dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **) &dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **) &dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << < 1, size >> > (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    */
    return 0;
}