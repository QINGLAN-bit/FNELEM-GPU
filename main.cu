/**
FNELEM-GPU MAIN FILE
Performs finite element structural analysis using an 4-node membrane, matrix inversion
was calculated using a CUDA algorithm (Gauss Jordan inversion).

@author ppizarror
@date 19/11/2018
@license
	MIT License
	Copyright (c) 2018 Pablo Pizarro R.

	Permission is hereby granted, free of charge, to any person obtaining a copy
	of this software and associated documentation files (the "Software"), to deal
	in the Software without restriction, including without limitation the rights
	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
	copies of the Software, and to permit persons to whom the Software is
	furnished to do so, subject to the following conditions:

	The above copyright notice and this permission notice shall be included in all
	copies or substantial portions of the Software.

	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
	SOFTWARE.
*/

// CUDA library imports
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>

// FNELEM library imports
#include "fnelem/math/fematrix.cpp"
#include "fnelem/math/fematrix_utils.cpp"
#include "fnelem/math/matrix_inversion_cpu.cpp"
#include "fnelem/math/matrix_inversion_cuda.cu"

#include "fnelem/analysis/static_analysis.cpp"
#include "fnelem/model/base/model.cpp"
#include "fnelem/model/base/model_component.cpp"
#include "fnelem/model/elements/element.cpp"
#include "fnelem/model/elements/membrane.cpp"
#include "fnelem/model/loads/load.cpp"
#include "fnelem/model/loads/load_membrane_distributed.cpp"
#include "fnelem/model/loads/load_node.cpp"
#include "fnelem/model/loads/load_pattern.cpp"
#include "fnelem/model/loads/load_pattern_constant.cpp"
#include "fnelem/model/nodes/node.cpp"
#include "fnelem/model/restraints/restraint.cpp"
#include "fnelem/model/restraints/restraint_node.cpp"

#include "test/test_suite.h"

int main() {

    // test_suite(); // Test all
    test_analysis(); // Test analysis

    return 0;
}