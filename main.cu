/**
FNELEM-GPU MAIN FILE
Performs finite element structural analysis using an 4-node membrane, matrix inversion
was calculated using a CUDA algorithm (Gauss Jordan inversion).

@author ppizarror
@date 19/11/2018
@license
	MIT License
	Copyright (c) 2018 Pablo Pizarro R.

	Permission is hereby granted, free of charge, to any person obtaining a copy
	of this software and associated documentation files (the "Software"), to deal
	in the Software without restriction, including without limitation the rights
	to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
	copies of the Software, and to permit persons to whom the Software is
	furnished to do so, subject to the following conditions:

	The above copyright notice and this permission notice shall be included in all
	copies or substantial portions of the Software.

	THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
	IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
	FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
	AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
	LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
	OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
	SOFTWARE.
*/

// CUDA library imports
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>

// FNELEM library imports
#include "fnelem/math/fematrix.cpp"
#include "fnelem/math/fematrix_utils.cpp"
#include "fnelem/math/matrix_inversion_cpu.cpp"
#include "fnelem/math/matrix_inversion_cuda.cu"
#include "fnelem/model/node/node.h"

int main() {

    // Create nodes
    std::vector<Node> nodes = std::vector<Node>();

    // Creates matrix
    FEMatrix mat = FEMatrix(3, 3);
    mat.set(0, 0, 1);
    mat.set(0, 1, 2);
    mat.set(0, 2, 3);
    mat.set(1, 0, 5);
    mat.set(1, 1, 2);
    mat.set(1, 2, 1);
    mat.set(2, 0, 2);
    mat.set(2, 1, 2);
    mat.set(2, 2, 3);

    FEMatrix *imat = matrix_inverse_cuda(&mat);
    imat->disp();

    return 0;
}