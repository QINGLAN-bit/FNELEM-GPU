/**
FNELEM-GPU MAIN FILE
Performs finite element structural analysis using an 4-node membrane, matrix inversion
was calculated using a CUDA algorithm (Gauss Jordan inversion).

@package fnelem.analysis
@author ppizarror
@date 19/11/2018
@license
    MIT License
    Copyright (c) 2018 Pablo Pizarro R.

    Permission is hereby granted, free of charge, to any person obtaining a copy
    of this software and associated documentation files (the "Software"), to deal
    in the Software without restriction, including without limitation the rights
    to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
    copies of the Software, and to permit persons to whom the Software is
    furnished to do so, subject to the following conditions:

    The above copyright notice and this permission notice shall be included in all
    copies or substantial portions of the Software.

    THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
    IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
    FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
    AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
    LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
    OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
    SOFTWARE.
*/

// CUDA library imports
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// FNELEM library imports
#include "fnelem/analysis/matrix_inversion.cu"
#include "fnelem/analysis/matrix_utils.cpp"

int main() {

    // Creates matrix
    const int n = 3;
    double *L = new double[n * n];

    L[0 * 3 + 0] = 1;
    L[0 * 3 + 1] = 2;
    L[0 * 3 + 2] = 3;
    L[1 * 3 + 0] = 5;
    L[1 * 3 + 1] = 2;
    L[1 * 3 + 2] = 1;
    L[2 * 3 + 0] = 2;
    L[2 * 3 + 1] = 2;
    L[2 * 3 + 2] = 3;
    double *iL = inverse_matrix(L, 3);
    save_matrix_to_file(iL, "inverse.txt", n, n);

    return 0;
}